#include "cuda_vector.cuh"

cudaVector::cudaVector(unsigned int n) : mData{nullptr}, mSize{n} {
	if(hipError_t err = hipMalloc(&mData, sizeof(float) * mSize); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}
}
