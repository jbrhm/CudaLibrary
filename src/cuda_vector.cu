#include "cuda_vector.cuh"

cudaVector::cudaVector(unsigned int n, float* data) : mData{nullptr}, mSize{n} {
	if(hipError_t err = hipMalloc(&mData, sizeof(float) * mSize); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}

	if(hipError_t err = hipMemcpy(mData, data, sizeof(float) * mSize, hipMemcpyHostToDevice); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}
}

cudaVector::~cudaVector(){
	hipFree(mData);
}
