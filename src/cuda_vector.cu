#include "hip/hip_runtime.h"
#include "cuda_vector.cuh"

template<unsigned int READ_SIZE>
__global__ void vector_add(float* data1, float* data2, float* out, unsigned int n){
	int id = (blockIdx.x * blockDim.x + threadIdx.x) * READ_SIZE;

	unsigned int upperBound = (n <= (id + READ_SIZE)) ? n : id + READ_SIZE;

	for(unsigned int i = id; i < upperBound; ++i){
		out[i] = data1[i] + data2[i];
	}
}

cudaVector::cudaVector(unsigned int n, float* data) : mData{nullptr}, mSize{n} {
	if(hipError_t err = hipMalloc(&mData, sizeof(float) * mSize); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}

	if(hipError_t err = hipMemcpy(mData, data, sizeof(float) * mSize, hipMemcpyHostToDevice); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}
}

void cudaVector::syncHost(float* hostData){
	if(hipError_t err = hipMemcpy(hostData, mData, sizeof(float) * mSize, hipMemcpyDeviceToHost); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}
}

void cudaVector::syncDevice(float* hostData){
	if(hipError_t err = hipMemcpy(hostData, mData, sizeof(float) * mSize, hipMemcpyHostToDevice); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}
}
void cudaVector::vectorAdd(cudaVector* vec1, cudaVector* vec2, cudaVector* out){
	unsigned int BLOCK_SIZE = 32;
	unsigned int GRID_SIZE = std::ceil(static_cast<float>(vec1->mSize)/(READ_SIZE * BLOCK_SIZE));

	if(vec1->mSize != vec2->mSize || vec2->mSize != out->mSize){
		throw std::runtime_error("Vector Sizes Do Not Match");
	}

	vector_add<READ_SIZE><<<GRID_SIZE, BLOCK_SIZE>>>(vec1->mData, vec2->mData, out->mData, vec1->mSize);
}

cudaVector::~cudaVector(){
	hipFree(mData);
}

