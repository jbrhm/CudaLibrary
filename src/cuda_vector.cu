#include "cuda_vector.cuh"

cudaVector::cudaVector(unsigned int n, float* data) : mData{nullptr}, mSize{n} {
	if(hipError_t err = hipMalloc(&mData, sizeof(float) * mSize); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}

	if(hipError_t err = hipMemcpy(mData, data, sizeof(float) * mSize, hipMemcpyHostToDevice); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}
}

void cudaVector::syncHost(float* hostData){
	if(hipError_t err = hipMemcpy(hostData, mData, sizeof(float) * mSize, hipMemcpyDeviceToHost); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}
}

void cudaVector::syncDevice(float* hostData){
	if(hipError_t err = hipMemcpy(hostData, mData, sizeof(float) * mSize, hipMemcpyHostToDevice); err != hipSuccess){
		throw std::runtime_error(std::format("Cuda Failed: {}", std::string(hipGetErrorName(err))));
	}
}

cudaVector::~cudaVector(){
	hipFree(mData);
}
