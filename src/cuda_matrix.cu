#include "hip/hip_runtime.h"
#include "cuda_matrix.cuh"

__global__ void rowColProduct(double* dataA, double* dataB, double* dataC, unsigned int N){
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if(id < N * N){
		int row = id / N;
		int col = id % N;

		double sum = 0;

		for(unsigned int i = 0; i < N; ++i){
			sum += dataA[row * N + i] * dataB[col + N * i];
		}

		dataC[row * N + col] = sum;
	}
}

cudaMatrix::cudaMatrix(unsigned int N, double* data) : mN{N}, mData{nullptr} {
	if(hipError_t err = hipMalloc(&mData, mN * mN * sizeof(double)); err != hipSuccess) std::cout << hipGetErrorString(err);
	hipMemcpy(mData, data, mN * mN * sizeof(double), hipMemcpyHostToDevice);
}

void cudaMatrix::syncHost(double* hostData){
	hipMemcpy(hostData, mData, mN * mN * sizeof(double), hipMemcpyDeviceToHost);
}

void cudaMatrix::multiply(cudaMatrix &matA, cudaMatrix &matB, cudaMatrix &matC){

	constexpr unsigned int THREADS = 32;

	if(matA.mN != matB.mN || matA.mN != matC.mN) throw std::runtime_error("Matrices are not the same size!");

	rowColProduct<<<std::ceil((matA.mN * matA.mN)/static_cast<double>(THREADS)), THREADS>>>(matA.mData, matB.mData, matC.mData, matA.mN);
}

cudaMatrix::~cudaMatrix(){
	hipFree(mData);
}
